#include "hip/hip_runtime.h"
// -*- C++ -*-
#include <cstdlib>
#include <cstdio>
#include <cstring>
#include <cmath>
#include <cassert>
#include <ctime>
#include <cfloat>

#include <hip/hip_runtime.h>

#include "ManyMatrixMultiplications_cuda.cuh"
#include "../GpuUtilities.h"

__global__
void
kernel_nextThreadNextEntry_serialMatrices(const unsigned long numberOfMatricesToMultiply,
                                          const unsigned long matrixSize,
                                          const double * const __restrict__ dev_leftMatrices,
                                          const double * const __restrict__ dev_rightMatrices,
                                          double * const __restrict__ dev_resultMatrices) {

  // TODO:

}

__global__
void
kernel_nextThreadNextMatrix_deepEntries(const unsigned long numberOfMatricesToMultiply,
                                        const unsigned long matrixSize,
                                        const double * const __restrict__ dev_leftMatrices,
                                        const double * const __restrict__ dev_rightMatrices,
                                        double * const __restrict__ dev_resultMatrices) {

  // TODO:

}

void
runGpuTimingTest(const unsigned int numberOfTrials,
                 const unsigned int maxNumberOfBlocks,
                 const unsigned int numberOfThreadsPerBlock,
                 const CudaManyMatrixMultiplicationStyle cudaManyMatrixMultiplicationStyle,
                 const unsigned int numberOfMatricesToMultiply,
                 const unsigned int matrixSize,
                 const double * leftMatrices,
                 const double * rightMatrices,
                 double * resultMatrices,
                 double * elapsedTime) {

  const unsigned int numberOfEntriesInAllMatrices =
    numberOfMatricesToMultiply * matrixSize * matrixSize;

  // allocate device-side matrices
  double * dev_leftMatrices;
  checkCudaError(hipMalloc((void **) &dev_leftMatrices,
                            numberOfEntriesInAllMatrices*sizeof(double)));
  checkCudaError(hipMemcpy(dev_leftMatrices, leftMatrices,
                            numberOfEntriesInAllMatrices*sizeof(double),
                            hipMemcpyHostToDevice));
  double * dev_rightMatrices;
  checkCudaError(hipMalloc((void **) &dev_rightMatrices,
                            numberOfEntriesInAllMatrices*sizeof(double)));
  checkCudaError(hipMemcpy(dev_rightMatrices, rightMatrices,
                            numberOfEntriesInAllMatrices*sizeof(double),
                            hipMemcpyHostToDevice));

  // allocate device-side result matrices
  double * dev_resultMatrices;
  checkCudaError(hipMalloc((void **) &dev_resultMatrices,
                            numberOfEntriesInAllMatrices*sizeof(double)));

  // calculate the number of blocks
  const unsigned int numberOfBlocks =
    min(maxNumberOfBlocks,
        (unsigned int)ceil(numberOfEntriesInAllMatrices /
                           double(numberOfThreadsPerBlock)));

  *elapsedTime = DBL_MAX; // sigh, no numeric_limits

  // run the test repeatedly
  for (unsigned int trialNumber = 0;
       trialNumber < numberOfTrials; ++trialNumber) {

    // this forces the GPU to run another kernel, kind of like
    //  "resetting the cache" for the cpu versions.
    GpuUtilities::resetGpu();

    // Wait for any kernels to stop
    checkCudaError(hipDeviceSynchronize());

    // Start timing
    const TimeUtility::PreCpp11TimePoint tic = TimeUtility::getCurrentTime();

    // run the kernel
    if (cudaManyMatrixMultiplicationStyle ==
        NextThreadNextEntry_serialMatrices) {
      kernel_nextThreadNextEntry_serialMatrices<<<numberOfBlocks,
        numberOfThreadsPerBlock>>>(numberOfMatricesToMultiply,
                                   matrixSize,
                                   dev_leftMatrices,
                                   dev_rightMatrices,
                                   dev_resultMatrices);
    } else if (cudaManyMatrixMultiplicationStyle ==
               NextThreadNextMatrix_deepEntries) {
      kernel_nextThreadNextMatrix_deepEntries<<<numberOfBlocks,
        numberOfThreadsPerBlock>>>(numberOfMatricesToMultiply,
                                   matrixSize,
                                   dev_leftMatrices,
                                   dev_rightMatrices,
                                   dev_resultMatrices);
    }

    // see if there was an error in the kernel launch
    checkCudaError(hipPeekAtLastError());

    // wait for the kernel to stop
    checkCudaError(hipDeviceSynchronize());

    // Stop timing
    const TimeUtility::PreCpp11TimePoint toc = TimeUtility::getCurrentTime();
    const double thisTrialsElapsedTime =
      TimeUtility::getElapsedTime(tic, toc);
    *elapsedTime = std::min(*elapsedTime, thisTrialsElapsedTime);
  }

  // copy back the output matrices
  checkCudaError(hipMemcpy(resultMatrices, dev_resultMatrices,
                            numberOfEntriesInAllMatrices*sizeof(double),
                            hipMemcpyDeviceToHost));

  // clean up
  checkCudaError(hipFree(dev_leftMatrices));
  checkCudaError(hipFree(dev_rightMatrices));
  checkCudaError(hipFree(dev_resultMatrices));
}
