#include "hip/hip_runtime.h"
// -*- C++ -*-
#include <cstdlib>
#include <cstdio>
#include <cstring>
#include <cmath>
#include <cassert>
#include <cfloat>

#include <iostream>

#include <hip/hip_runtime.h>

#include "BinarySearch_cuda.cuh"
#include "../GpuUtilities.h"

__global__
static void
findKeysInSortedNumbers_kernel(const unsigned int * __restrict__ sortedNumbers,
                               const unsigned int numberOfSortedNumbers,
                               const unsigned int * __restrict__ input,
                               const unsigned int inputSize,
                               bool * __restrict__ output) {

        for (unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
             i < inputSize; i += blockDim.x * gridDim.x) {

                unsigned first = 0, last = numberOfSortedNumbers - 1;
                auto key = input[i];

                output[i] = false;

                while (first <= last) {
                        unsigned midx = first + (last - first)/2;
                        auto mid = sortedNumbers[midx];
                        if (mid < key)
                                first = midx+1;
                        else if (mid > key)
                                last = midx-1;
                        else {
                                output[i] = true;
                                break;
                        }
                }
        }
}

void
runGpuTimingTest(const unsigned int numberOfTrials,
                 const unsigned int maxNumberOfBlocks,
                 const unsigned int numberOfThreadsPerBlock,
                 const unsigned int * sortedNumbers,
                 const unsigned int numberOfSortedNumbers,
                 const unsigned int * input,
                 const unsigned int inputSize,
                 bool * output,
                 double * elapsedTime) {

  // allocate device-side inputs
  unsigned int * dev_sortedNumbers;
  checkCudaError(hipMalloc((void **) &dev_sortedNumbers,
                            numberOfSortedNumbers*sizeof(unsigned int)));
  unsigned int * dev_input;
  checkCudaError(hipMalloc((void **) &dev_input,
                            inputSize*sizeof(unsigned int)));

  // copy host inputs to device
  checkCudaError(hipMemcpy(dev_sortedNumbers, sortedNumbers,
                            numberOfSortedNumbers*sizeof(unsigned int),
                            hipMemcpyHostToDevice));
  checkCudaError(hipMemcpy(dev_input, input,
                            inputSize*sizeof(unsigned int),
                            hipMemcpyHostToDevice));

  // allocate device-side outputs
  bool * dev_output;
  checkCudaError(hipMalloc((void **) &dev_output,
                            inputSize*sizeof(bool)));

  // calculate the number of blocks
  const unsigned int numberOfBlocks =
    min(maxNumberOfBlocks,
        (unsigned int)ceil(inputSize/double(numberOfThreadsPerBlock)));

  *elapsedTime = DBL_MAX; // sigh, no numeric_limits

  // run the test repeatedly
  for (unsigned int trialNumber = 0;
       trialNumber < numberOfTrials; ++trialNumber) {

          std::cout << __func__ << "trialNumber=" << trialNumber
                    << " maxNumberOfBlocks=" << maxNumberOfBlocks
                    << " numberOfThreadsPerBlock=" << numberOfThreadsPerBlock
                    << " numberOfSortedNumbers=" << numberOfSortedNumbers
                    << std::endl;
          
    // this forces the GPU to run another kernel, kind of like
    //  "resetting the cache" for the cpu versions.
    GpuUtilities::resetGpu();

    // Wait for any kernels to stop
    checkCudaError(hipDeviceSynchronize());

    // Start timing
    const TimeUtility::PreCpp11TimePoint tic = TimeUtility::getCurrentTime();

    // run kernel
    findKeysInSortedNumbers_kernel<<<numberOfBlocks,
      numberOfThreadsPerBlock>>>(dev_sortedNumbers,
                                 numberOfSortedNumbers,
                                 dev_input,
                                 inputSize,
                                 dev_output);
    // see if there was an error in the kernel launch
    checkCudaError(hipPeekAtLastError());

    // wait for the kernel to stop
    checkCudaError(hipDeviceSynchronize());

    // Stop timing
    const TimeUtility::PreCpp11TimePoint toc = TimeUtility::getCurrentTime();
    const double thisTrialsElapsedTime =
      TimeUtility::getElapsedTime(tic, toc);
    *elapsedTime = std::min(*elapsedTime, thisTrialsElapsedTime);
  }

  // copy device outputs back to host
  checkCudaError(hipMemcpy(output, dev_output, inputSize*sizeof(bool),
                            hipMemcpyDeviceToHost));

  checkCudaError(hipFree(dev_sortedNumbers));
  checkCudaError(hipFree(dev_input));
  checkCudaError(hipFree(dev_output));
}
